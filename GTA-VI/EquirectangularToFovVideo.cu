#include "hip/hip_runtime.h"
// EquirectangularToFovVideo.cu

#include "EquirectangularToFovVideo.h"
#include "Util.h"

#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cmath>

#define PI 3.14159265

using namespace std;

// PUBLIC:

EquirectangularToFovVideo::EquirectangularToFovVideo(Arff* pArff) : EquirectangularToFovBase(pArff), eq_d{nullptr}, fov_d{nullptr}, m_pInfo{new ConvertionInfo()}
{
}

/*virtual*/ EquirectangularToFovVideo::~EquirectangularToFovVideo()
{
    if (eq_d != nullptr)
        hipFree(eq_d);
    if (fov_d != nullptr)
        hipFree(fov_d);
    delete m_pInfo;
}

// The following functions are the same as the ones provided from the base class
// but specialized to run on the device (GPU)

__device__ void EquirectangularToSpherical_d(unsigned int xEq, unsigned int yEq, unsigned int widthPx, unsigned int heightPx, double *horRads, double *verRads)
{
    *horRads = (xEq * 2.0 * PI) / widthPx;
    *verRads = (yEq * PI) / heightPx;
}

__device__ void SphericalToEquirectangular_d(double horRads, double verRads, unsigned int widthPx, unsigned int heightPx, unsigned int *xEq, unsigned int *yEq)
{
    int x = (int)((horRads / (2.0 * PI)) * widthPx + 0.5); // round double to closer int
    int y = (int)((verRads / PI) * heightPx + 0.5);

    // make sure returned values are within the video
    if (x < 0)
        x = widthPx + x - 1;
    else if (x >= (int)widthPx)
        x -= widthPx;
    *xEq = (unsigned int) x;

    if (y < 0)
        y = heightPx + y - 1;

    if (y >= (int)heightPx)
        y = 2 * heightPx - y - 1;

    *yEq = (unsigned int) y;
}

__device__ void SphericalToCartesian_d(double horRads, double verRads, Vec3 *cart)
{
    cart->x = sin(verRads)*cos(horRads);
    cart->y = cos(verRads);
    cart->z = sin(verRads)*sin(horRads);
}

__device__ void CartesianToSpherical_d(Vec3 cart, double *horRads, double *verRads)
{
    *horRads = atan2(cart.z, cart.x);
    *verRads = acos(cart.y);
}

__device__ Vec3 RotatePoint_d(Matrix33 rot, Vec3 v)
{
    //Vec3 res(0,0,0);
    Vec3 res = v; // Avoid __device__ constructor by using copy constructor

    res.x = rot.mat[0][0]*v.x + rot.mat[0][1]*v.y + rot.mat[0][2]*v.z;
    res.y = rot.mat[1][0]*v.x + rot.mat[1][1]*v.y + rot.mat[1][2]*v.z;
    res.z = rot.mat[2][0]*v.x + rot.mat[2][1]*v.y + rot.mat[2][2]*v.z;

    return res;
}

__global__ void GPUCalculation(const uchar *eq_d, uchar *fov_d, ConvertionInfo *info)
{
    double fovWidth_rads = (info->fovWidth_deg * PI / 180);
    double fovHeight_rads = (info->fovHeight_deg * PI / 180);

    double horRads, verRads;
    double vidHorRads, vidVerRads;
    unsigned int xEq, yEq;

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    int x = idx;
    int y = idy;

    if (x >= info->fovWidth_px || y >= info->fovHeight_px)
        return;

    horRads =  x * fovWidth_rads / info->fovWidth_px - fovWidth_rads / 2.0;
    verRads = y * fovHeight_rads / info->fovHeight_px - fovHeight_rads / 2.0;
    // make it point towards center of equirectangular projection
    horRads += PI;
    verRads += PI/2;
    //Vec3 pixelVec(0,0,0);
    Vec3 pixelVec = info->tmpVec; // Avoid __device__ constructor by using copy constructor
    SphericalToCartesian_d(horRads, verRads, &pixelVec);

    Vec3 vidPixelVec = RotatePoint_d(info->rot, pixelVec);
    CartesianToSpherical_d(vidPixelVec, &vidHorRads, &vidVerRads);
    SphericalToEquirectangular_d(vidHorRads, vidVerRads, info->eqWidth_px, info->eqHeight_px, &xEq, &yEq);

	int posEq = yEq*info->eqWidth_px*4 + xEq*4;
    int posFov = y*info->fovWidth_px*4 + x*4;
    *(fov_d + posFov) = *(eq_d + posEq);
    *(fov_d + posFov + 1) = *(eq_d + posEq + 1);
    *(fov_d + posFov + 2) = *(eq_d + posEq + 2);
    *(fov_d + posFov + 3) = *(eq_d + posEq + 3);
}

bool EquirectangularToFovVideo::Convert(const QImage *eqImage, long int time, QImage *fovImage)
{
    double xEqHead, yEqHead, tiltHead; 
    GetHeadPos(time, &xEqHead, &yEqHead, &tiltHead);

    double horHeadRads, verHeadRads;
    EquirectangularToSpherical(xEqHead, yEqHead, m_pArff->WidthPx(), m_pArff->HeightPx(), &horHeadRads, &verHeadRads);

    Vec3 headVec(0,0,0);
    SphericalToCartesian(horHeadRads, verHeadRads, &headVec);
    
    Vec3 vidVec(-1,0,0); // pointing to the middle of equirectangular projection
    double headTiltRads = tiltHead * PI / 180;

    Matrix33 rot = HeadToVideoRotation(headVec, headTiltRads, vidVec);

    const uchar *eqImageBits = eqImage->bits();
    uchar *fovImageBits = fovImage->bits();

    GenerateSampling(fovImage);

    // Set up GPU for calculation
    ConvertionInfo *info = new ConvertionInfo();
    ConvertionInfo *info_d;
    m_pInfo->rot = rot;
    m_pInfo->fovWidth_deg = m_fovWidthDeg;
    m_pInfo->fovHeight_deg = m_fovHeightDeg;
    m_pInfo->fovWidth_px = fovImage->width();
    m_pInfo->fovHeight_px = fovImage->height();
    m_pInfo->eqWidth_px = eqImage->width();
    m_pInfo->eqHeight_px = eqImage->height();

    if (eq_d == nullptr)
        hipMalloc((void**)&eq_d, eqImage->byteCount());
    if (fov_d == nullptr)
        hipMalloc((void**)&fov_d, fovImage->byteCount());
    hipMalloc((void**)&info_d, sizeof(ConvertionInfo)); 
    hipMemcpy(eq_d, eqImageBits, eqImage->byteCount(), hipMemcpyHostToDevice);
    hipMemcpy(fov_d, fovImageBits, fovImage->byteCount(), hipMemcpyHostToDevice);
    hipMemcpy(info_d, m_pInfo, sizeof(ConvertionInfo), hipMemcpyHostToDevice);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int maxThreads = devProp.maxThreadsPerBlock;
    int maxThreadsDim = floor(sqrt(maxThreads));

    dim3 dimBlock(maxThreadsDim, maxThreadsDim);
    dim3 dimGrid(fovImage->width()/dimBlock.x + 1, fovImage->height()/dimBlock.y + 1);

    GPUCalculation<<<dimGrid,dimBlock>>>(eq_d, fov_d, info_d);

    hipMemcpy(fovImageBits, fov_d, fovImage->byteCount(), hipMemcpyDeviceToHost);
    hipFree(info_d);

    // *** Placeholder
    return true;
}

double EquirectangularToFovVideo::GetAspectRatio()
{
    return (double)m_fovWidthPx/m_fovHeightPx;
}

// PRIVATE:

void EquirectangularToFovVideo::GenerateSampling(const QImage *image)
{
    if ((int)m_vHorSampling.size() == image->width() && (int)m_vVerSampling.size() == image->height())
        return;

    m_vHorSampling.resize(image->width());
    m_vVerSampling.resize(image->height());

    double fovWidthRads = (m_fovWidthDeg * PI / 180);
    double fovHeightRads = (m_fovHeightDeg * PI / 180);

    Generate1DSampling(fovWidthRads, &m_vHorSampling);
    Generate1DSampling(fovHeightRads, &m_vVerSampling);
}

void EquirectangularToFovVideo::Generate1DSampling(double fovRads, vector<double> *samples)
{
    for (size_t i=0; i<samples->size(); i++)
        (*samples)[i] = i * fovRads / samples->size() - fovRads / 2.0;
}
